#include "hip/hip_runtime.h"
/*
 * NWOR Draft Commit Kernel Implementation
 *
 * This kernel scatters accepted tokens from staged draft buffers to the KV cache.
 * It reuses the exact vectorization and quantization logic from reshape_and_cache_flash_kernel
 * to ensure correctness and performance.
 */

#include "nwor_commit.h"

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_utils.h"
#include "cuda_compat.h"
#include "dispatch_utils.h"
#include "quantization/vectorization_utils.cuh"

#ifdef USE_ROCM
  #include "quantization/w8a8/fp8/amd/quant_utils.cuh"
#else
  #include "quantization/w8a8/fp8/nvidia/quant_utils.cuh"
#endif

namespace vllm {

// Commit draft kernel - copied vectorization from reshape_and_cache_flash_kernel
// Key differences:
// 1. Early exit on mask[token_idx] == false (Issue #3: mask early-return)
// 2. No atomic count (Issue #5: count computed in Python)
template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void commit_draft_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // Layout-dependent, see head_stride check
    cache_t* __restrict__ value_cache,
    const bool* __restrict__ mask,             // [num_tokens]
    const int32_t* __restrict__ slot_mapping,  // [num_tokens] - guaranteed int32
    const float* k_scale,
    const float* v_scale,
    const bool scale_is_per_token,
    const int64_t key_stride,
    const int64_t value_stride,
    const int64_t block_stride,
    const int64_t page_stride,
    const int64_t head_stride,
    const int num_heads,
    const int head_size,
    const int block_size
) {
    const int64_t token_idx = blockIdx.x;

    // Issue #3: Mask early-return BEFORE any other work (avoid divergence)
    if (!mask[token_idx]) {
        return;
    }

    const int64_t slot_idx = slot_mapping[token_idx];
    // NOTE: slot_idx can be -1 if the token is padded
    if (slot_idx < 0) {
        return;
    }

    const int64_t block_idx = slot_idx / block_size;
    const int64_t block_offset = slot_idx % block_size;
    const int n_elems = num_heads * head_size;

    // Pointers to the beginning of the source row for this token
    const scalar_t* __restrict__ key_src = key + token_idx * key_stride;
    const scalar_t* __restrict__ value_src = value + token_idx * value_stride;

    // Find the start position inside the kv-cache for this token
    cache_t* __restrict__ key_dst =
        key_cache + block_idx * block_stride + block_offset * page_stride;
    cache_t* __restrict__ value_dst =
        value_cache + block_idx * block_stride + block_offset * page_stride;

    // This is true for the NHD layout where `head_stride == head_size`
    const bool is_contiguous_heads = (head_stride == head_size);

    // Issue #3: Quantization scale handling (per-token vs scalar)
    float k_scale_val = 0.f;
    float v_scale_val = 0.f;
    if constexpr (kv_dt != Fp8KVCacheDataType::kAuto) {
        if (k_scale != nullptr) {
            k_scale_val = scale_is_per_token ? k_scale[token_idx] : k_scale[0];
        }
        if (v_scale != nullptr) {
            v_scale_val = scale_is_per_token ? v_scale[token_idx] : v_scale[0];
        }
    }

    // Issue #4: Exact vectorization copied from reshape_and_cache_flash_kernel
    constexpr int VEC_SIZE = (sizeof(scalar_t) == 2) ? 8 : 4;
    CopyWithScaleOp<cache_t, scalar_t, kv_dt> k_op{k_scale_val};
    CopyWithScaleOp<cache_t, scalar_t, kv_dt> v_op{v_scale_val};

    if (is_contiguous_heads) {
        // NHD layout: [num_blocks, block_size, num_heads, head_size]
        vectorize_with_alignment<VEC_SIZE>(key_src, key_dst, n_elems, threadIdx.x,
                                           blockDim.x, k_op);

        vectorize_with_alignment<VEC_SIZE>(value_src, value_dst, n_elems,
                                           threadIdx.x, blockDim.x, v_op);

    } else {
        // HND layout: [num_blocks, num_heads, block_size, head_size]
        // Heads are strided, but each head_size segment is contiguous
        const int lane = threadIdx.x & 31;     // 0..31 within warp
        const int warp_id = threadIdx.x >> 5;  // warp index within block
        const int warps_per_block = blockDim.x >> 5;

        for (int head = warp_id; head < num_heads; head += warps_per_block) {
            const scalar_t* __restrict__ k_src_h = key_src + head * head_size;
            const scalar_t* __restrict__ v_src_h = value_src + head * head_size;

            cache_t* __restrict__ k_dst_h =
                key_dst + static_cast<int64_t>(head) * head_stride;
            cache_t* __restrict__ v_dst_h =
                value_dst + static_cast<int64_t>(head) * head_stride;

            // Within each head, let the 32 threads of the warp perform the vector copy
            vectorize_with_alignment<VEC_SIZE>(k_src_h, k_dst_h, head_size, lane, 32,
                                               k_op);

            vectorize_with_alignment<VEC_SIZE>(v_src_h, v_dst_h, head_size, lane, 32,
                                               v_op);
        }
    }
}

}  // namespace vllm

// Template dispatch macro matching existing pattern
#define CALL_COMMIT_DRAFT_KERNEL(KV_T, CACHE_T, KV_DTYPE)               \
  vllm::commit_draft_kernel<KV_T, CACHE_T, KV_DTYPE>                    \
      <<<grid, block, 0, stream>>>(                                     \
          reinterpret_cast<const KV_T*>(key_ptr),                       \
          reinterpret_cast<const KV_T*>(value_ptr),                     \
          reinterpret_cast<CACHE_T*>(key_cache_ptr),                    \
          reinterpret_cast<CACHE_T*>(value_cache_ptr),                  \
          reinterpret_cast<const bool*>(mask_ptr),                      \
          reinterpret_cast<const int32_t*>(slot_ptr),                   \
          k_scale_ptr ? reinterpret_cast<const float*>(k_scale_ptr) : nullptr, \
          v_scale_ptr ? reinterpret_cast<const float*>(v_scale_ptr) : nullptr, \
          scale_is_per_token,                                           \
          key_stride, value_stride, block_stride, page_stride,          \
          head_stride, static_cast<int>(num_heads), static_cast<int>(head_size), static_cast<int>(block_size));

// Main entry point with full validation and dispatch
void commit_draft_layer(
    int64_t key_ptr,
    int64_t value_ptr,
    int64_t key_cache_ptr,
    int64_t value_cache_ptr,
    int64_t mask_ptr,
    int64_t slot_ptr,
    int64_t k_scale_ptr,
    int64_t v_scale_ptr,
    bool scale_is_per_token,
    int64_t num_tokens,
    int64_t num_heads,
    int64_t head_size,
    int64_t block_size,
    int64_t block_stride,
    int64_t page_stride,
    int64_t head_stride,
    int64_t layout,
    const std::string& key_value_dtype,
    const std::string& kv_cache_dtype
) {
    // Issue #7: TORCH_CHECK for null pointers
    TORCH_CHECK(key_ptr != 0, "key_ptr is null");
    TORCH_CHECK(value_ptr != 0, "value_ptr is null");
    TORCH_CHECK(key_cache_ptr != 0, "key_cache_ptr is null");
    TORCH_CHECK(value_cache_ptr != 0, "value_cache_ptr is null");
    TORCH_CHECK(mask_ptr != 0, "mask_ptr is null");
    TORCH_CHECK(slot_ptr != 0, "slot_ptr is null");
    TORCH_CHECK(num_tokens > 0, "num_tokens must be positive");
    TORCH_CHECK(num_heads > 0, "num_heads must be positive");
    TORCH_CHECK(head_size > 0, "head_size must be positive");
    TORCH_CHECK(block_size > 0, "block_size must be positive");

    // Compute strides for draft tensors
    // Key/value layout: [num_tokens, num_heads, head_size]
    int64_t key_stride = num_heads * head_size;
    int64_t value_stride = num_heads * head_size;

    // Issue #4: Grid/block dimensions matching reshape_and_cache_flash
    // Cast int64_t to unsigned int for dim3 constructor
    dim3 grid(static_cast<unsigned int>(num_tokens));
    dim3 block(static_cast<unsigned int>(std::min(num_heads * head_size, static_cast<int64_t>(512))));

    // Get CUDA stream
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // Issue #3: Convert key_value_dtype string to ScalarType
    at::ScalarType src_dtype;
    if (key_value_dtype == "fp16") {
        src_dtype = at::ScalarType::Half;
    } else if (key_value_dtype == "bf16") {
        src_dtype = at::ScalarType::BFloat16;
    } else if (key_value_dtype == "fp32") {
        src_dtype = at::ScalarType::Float;
    } else {
        TORCH_CHECK(false, "Unsupported key_value_dtype: ", key_value_dtype);
    }

    // Issue #3: Full dtype/cache type dispatch
    DISPATCH_BY_KV_CACHE_DTYPE(src_dtype, kv_cache_dtype, CALL_COMMIT_DRAFT_KERNEL);
}
